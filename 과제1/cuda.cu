#include "hip/hip_runtime.h"
﻿#include <iostream>
using namespace std;

#include <hip/hip_runtime.h>       
#include <stdlib.h>
#include <stdio.h>
#include <time.h>


#define THREADS pow(2,6)
#define BLOCKS pow(2,21)
#define NUM_VALS THREADS*BLOCKS

void print_elapsed(clock_t start, clock_t stop)
{
    double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
    return (float)rand() / (float)RAND_MAX;
}

void array_print(float* arr, int length)
{
    int i;
    for (i = 0; i < length; ++i) {
        printf("%1.3f ", arr[i]);
    }
    printf("\n");
}

void array_fill(float* arr, float* arr2, int* arr3, int length)
{
    srand(time(NULL));
    int i;
    for (i = 0; i < length; ++i) {
        arr[i] = random_float();
        arr2[i] = arr[i];
        arr3[i] = int(arr[i]);
    }
}

void array_fill(int* arr, int length)
{
    srand(time(NULL));
    int i;
    for (i = 0; i < length; ++i) {
        arr[i] = random_float();
    }
}


__global__ void bitonic_sort_step(float* dev_values, int j, int k)
{
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;//blockIdx.x로 복사본 구분

    ixj = i ^ j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort ascending */
            if (dev_values[i] > dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i & k) != 0) {
            /* Sort descending */
            if (dev_values[i] < dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

void bitonic_sort(float* values)
{
    float* dev_values;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void**)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    // <<<>>>>의 첫번째 인자에 3차원까지 지정가능 이것은 2차원인데 1이므로
    // BLOCKS*1 =BLOCKS수만큼들어감
    dim3 blocks(BLOCKS, 1);    /* Number of blocks   */
    dim3 threads(THREADS, 1);  /* Number of threads  */

    int j, k;
    /* Major step */
    for (k = 2; k <= NUM_VALS; k <<= 1) {
        /* Minor step */
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step << <blocks, threads >> > (dev_values, j, k);
            //blocks 만큼 복사본 생성(할당) 되고, 각각 block에 Threads개의 스레드가 실행함
            //총 작업수는 그럼 몇이게? 쓰레드*블럭수 만큼
        }
    }
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}


//cpu
void compAndSwap(int a[], int i, int j, int dir)
{
    if (dir == (a[i] > a[j]))
        swap(a[i], a[j]);
}

void bitonicMerge(int a[], int low, int cnt, int dir)
{
    if (cnt > 1)
    {
        int k = cnt / 2;
        for (int i = low; i < low + k; i++)
            compAndSwap(a, i, i + k, dir);
        bitonicMerge(a, low, k, dir);
        bitonicMerge(a, low + k, k, dir);
    }
}
void bitonicSort(int a[], int low, int cnt, int dir) {
    if (cnt > 1) {
        int k = cnt / 2;
        bitonicSort(a, low, k, 1);
        bitonicSort(a, low + k, k, 0);
        bitonicMerge(a, low, cnt, dir);
    }
}

#include<stdio.h>
#include<time.h>
#include<stdlib.h>

#pragma comment (linker,"/STACK:22496080")

void swap(int* arr, int a, int b) { // a, b swap 함수 구현
    float temp = arr[a];
    arr[a] = arr[b];
    arr[b] = temp;
}

void QuickSort(int* arr, int l, int r) {
    int i, j, mid, pivot;

    mid = (l + r) / 2; // pivot 중간값

    if (arr[l] > arr[mid]) // left가 중간값보다 크면
        swap(arr, l, mid); // left와 중간값 교체
    if (arr[l] > arr[r]) // left가 right보다 크면
        swap(arr, l, r); // left와 right 교체
    if (arr[mid] > arr[r]) // 중간값이 right보다 크면
        swap(arr, mid, r); // 중간값과 right 교체

    if (r - l + 1 > 3) { // 중간값 quick sort는 
                    // 우선적으로 3게의 값들에 대해 정렬을 수행
                    // 만약 정렬 대상이 3개 이하라면 위에서 이미 정렬이 끝남
                    // 따라서 데이터의 개수가 3개 초과일 때만 추가적으로 정렬 수행
        pivot = arr[mid];
        swap(arr, mid, r - 1);
        i = l;
        j = r - 1;

        while (1) { // 정렬 확인하고, left가 right보다 크면 swap
            while (arr[++i] < pivot && i < r);
            while (arr[--j] > pivot && l < j);
            if (i >= j) break;
            swap(arr, i, j);
        }

        swap(arr, i, r - 1);
        QuickSort(arr, l, i - 1);
        QuickSort(arr, i + 1, r);

    }
}

// gap만큼 떨어진 요소들을 삽입 정렬
// 정렬의 범위는 first에서 last까지
void inc_insertion_sort(float* arr, int first, int last, int gap) {
    int i, j, key;

    for (i = first + gap; i <= last; i = i + gap) {
        key = arr[i]; // 현재 삽입될 숫자인 i번째 정수를 key 변수로 복사

        // 현재 정렬된 배열은 i-gap까지이므로 i-gap번째부터 역순으로 조사한다.
        // j 값은 first 이상이어야 하고
        // key 값보다 정렬된 배열에 있는 값이 크면 j번째를 j+gap번째로 이동
        for (j = i - gap; j >= first && arr[j] > key; j = j - gap) {
            arr[j + gap] = arr[j]; // 레코드를 gap만큼 오른쪽으로 이동
        }

        arr[j + gap] = key;
    }
}

// 셸 정렬
void shell_sort(float* arr, int n) {
    int i, gap;

    for (gap = n / 2; gap > 0; gap = gap / 2) {
        if ((gap % 2) == 0) {
            gap++; // gap을 홀수로 만든다.
        }

        // 부분 리스트의 개수는 gap과 같다.
        for (i = 0; i < gap; i++) {
            // 부분 리스트에 대한 삽입 정렬 수행
            inc_insertion_sort(arr, i, n - 1, gap);
        }
    }
}


int main(void) {
    clock_t start, stop;
    float* values = (float*)malloc(NUM_VALS * sizeof(float));
    float* values2 = (float*)malloc(NUM_VALS * sizeof(float));
    int* values3 = (int*)malloc(NUM_VALS * sizeof(int));
    array_fill(values, values2, values3, NUM_VALS);

    start = clock();
    bitonic_sort(values);
    stop = clock();
    cout << "cuda ";
    print_elapsed(start, stop);

    start = clock();
    shell_sort(values2, NUM_VALS);
    stop = clock();
    cout << "shell sort ";
    print_elapsed(start, stop);


    start = clock();
    QuickSort(values3, 0, NUM_VALS-1);
    stop = clock();
    cout << "quick sort ";
    print_elapsed(start, stop);
}